
#include <hip/hip_runtime.h>






/*!
 * Compute the next power of 2 which occurs after a number.
 *
 * @param n
 */
__device__
int nextPower2(int n)
{
   int pow2 = 2;
   while ( pow2 < n )
   {
      pow2 *= 2;
   }

   return pow2;
}






/*!
 * Swap two values
 *
 * @param a
 * @param b
 */
__device__
void swap(float *a, float *b)
{
   float c = *a;
   *a = *b;
   *b = c;
}






/*!
 * Sort an array using bitonic sort. The array should have a size which is a
 * power of two.
 *
 * @param array
 * @param size
 * @param stride
 */
__device__
void bitonicSort(float *array, int size, int stride)
{
   int bsize = size / 2;
   int dir, a, b, t;

   for ( int ob = 2; ob <= size; ob *= 2 )
   {
      for ( int ib = ob; ib >= 2; ib /= 2 )
      {
         t = ib/2;
         for ( int i = 0; i < bsize; ++i )
         {
            dir = -((i/(ob/2)) & 0x1);
            a = (i/t) * ib + (i%t);
            b = a + t;

            a *= stride;
            b *= stride;

            if ( (!dir && (array[a] > array[b])) || (dir && (array[a] < array[b])) )
            {
               swap(&array[a], &array[b]);
            }
         }
      }
   }
}






/*!
 * Sort an array using bitonic sort, while also applying the same swap operations
 * to a second array of the same size. The arrays should have a size which is a
 * power of two.
 *
 * @param array
 * @param extra
 * @param size
 * @param stride
 */
__device__
void bitonicSortFF(float *array, float *extra, int size, int stride)
{
   int bsize = size / 2;
   int dir, a, b, t;

   for ( int ob = 2; ob <= size; ob *= 2 )
   {
      for ( int ib = ob; ib >= 2; ib /= 2 )
      {
         t = ib/2;
         for ( int i = 0; i < bsize; ++i )
         {
            dir = -((i/(ob/2)) & 0x1);
            a = (i/t) * ib + (i%t);
            b = a + t;

            a *= stride;
            b *= stride;

            if ( (!dir && (array[a] > array[b])) || (dir && (array[a] < array[b])) )
            {
               swap(&array[a], &array[b]);
               swap(&extra[a], &extra[b]);
            }
         }
      }
   }
}






/*!
 * Compute the rank of a sorted vector in place. In the event of ties,
 * the ranks are corrected using fractional ranking.
 *
 * @param array
 * @param n
 * @param stride
 */
__device__
void computeRank(float *array, int n, int stride)
{
   int i = 0;

   while ( i < n - 1 )
   {
      float a_i = array[i * stride];

      if ( a_i == array[(i + 1) * stride] )
      {
         int j = i + 2;
         int k;
         float rank = 0;

         // we have detected a tie, find number of equal elements
         while ( j < n && a_i == array[j * stride] )
         {
            ++j;
         }

         // compute rank
         for ( k = i; k < j; ++k )
         {
            rank += k;
         }

         // divide by number of ties
         rank /= (float) (j - i);

         for ( k = i; k < j; ++k )
         {
            array[k * stride] = rank;
         }

         i = j;
      }
      else
      {
         // no tie - set rank to natural ordered position
         array[i * stride] = i;
         ++i;
      }
   }

   if ( i == n - 1 )
   {
      array[(n - 1) * stride] = (float) (n - 1);
   }
}
