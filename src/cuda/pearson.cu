
#include <hip/hip_runtime.h>






/*!
 * Compute the Pearson correlation of a cluster in a pairwise data array.
 *
 * @param x
 * @param y
 * @param labels
 * @param sampleSize
 * @param stride
 * @param cluster
 * @param minSamples
 */
__device__
float Pearson_computeCluster(
   const float *x,
   const float *y,
   const char *labels,
   int sampleSize,
   int stride,
   char cluster,
   int minSamples)
{
   // compute intermediate sums
   int n = 0;
   float sumx = 0;
   float sumy = 0;
   float sumx2 = 0;
   float sumy2 = 0;
   float sumxy = 0;

   for ( int i = 0; i < sampleSize; ++i )
   {
      if ( labels[i * stride] == cluster )
      {
         float x_i = x[i];
         float y_i = y[i];

         sumx += x_i;
         sumy += y_i;
         sumx2 += x_i * x_i;
         sumy2 += y_i * y_i;
         sumxy += x_i * y_i;

         ++n;
      }
   }

   // compute correlation only if there are enough samples
   float result = NAN;

   if ( n >= minSamples )
   {
      result = (n*sumxy - sumx*sumy) / sqrt((n*sumx2 - sumx*sumx) * (n*sumy2 - sumy*sumy));
   }

   return result;
}






/*!
 * Compute the correlation of each cluster in a pairwise data array. The data array
 * should only contain the clean samples that were extracted from the expression
 * matrix, while the labels should contain all samples.
 *
 * @param numPairs
 * @param expressions
 * @param sampleSize
 * @param in_index
 * @param in_argsort
 * @param clusterSize
 * @param in_labels
 * @param minSamples
 * @param out_correlations
 */
__global__
void Pearson_compute(
   int numPairs,
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   const int *in_argsort,
   char clusterSize,
   const char *in_labels,
   int minSamples,
   float *out_correlations)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = gridDim.x * blockDim.x;

   if ( i >= numPairs )
   {
      return;
   }

   // initialize workspace variables
   int i_argsort = in_argsort[i];
   int2 index = in_index[i_argsort];
   const float *x = &expressions[index.x * sampleSize];
   const float *y = &expressions[index.y * sampleSize];
   const char *labels = &in_labels[i_argsort];
   float *correlations = &out_correlations[i_argsort];

   for ( char k = 0; k < clusterSize; ++k )
   {
      correlations[k * stride] = Pearson_computeCluster(x, y, labels, sampleSize, stride, k, minSamples);
   }
}
