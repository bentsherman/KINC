
#include <hip/hip_runtime.h>

// #include "linalg.cu"






/*!
 * Compute the initial labels for a gene pair in an expression matrix. Samples
 * with missing values and samples that fall below the expression threshold are
 * labeled as such, all other samples are labeled as cluster 0. The number of
 * clean samples is returned.
 *
 * @param numPairs
 * @param expressions
 * @param sampleSize
 * @param in_index
 * @param minExpression
 * @param out_N
 * @param out_labels
 */
__global__
void fetchPair(
   int numPairs,
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   int minExpression,
   int *out_N,
   char *out_labels)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = gridDim.x * blockDim.x;

   if ( i >= numPairs )
   {
      return;
   }

   // initialize variables
   int2 index = in_index[i];
   char *labels = &out_labels[i];
   int *p_N = &out_N[i];

   // index into gene expressions
   const float *x = &expressions[index.x * sampleSize];
   const float *y = &expressions[index.y * sampleSize];

   // label the pairwise samples
   int N = 0;

   for ( int i = 0, j = 0; i < sampleSize; i += 1, j += stride )
   {
      // label samples with missing values
      if ( isnan(x[i]) || isnan(y[i]) )
      {
         labels[j] = -9;
      }

      // label samples which fall below the expression threshold
      else if ( x[i] < minExpression || y[i] < minExpression )
      {
         labels[j] = -6;
      }

      // label any remaining samples as cluster 0
      else
      {
         N++;
         labels[j] = 0;
      }
   }

   // save number of clean samples
   *p_N = N;
}
